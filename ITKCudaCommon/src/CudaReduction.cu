/*=========================================================================
 *
 *  Copyright Insight Software Consortium
 *
 *  Licensed under the Apache License, Version 2.0 (the "License");
 *  you may not use this file except in compliance with the License.
 *  You may obtain a copy of the License at
 *
 *         http://www.apache.org/licenses/LICENSE-2.0.txt
 *
 *  Unless required by applicable law or agreed to in writing, software
 *  distributed under the License is distributed on an "AS IS" BASIS,
 *  WITHOUT WARRANTIES OR CONDITIONS OF ANY KIND, either express or implied.
 *  See the License for the specific language governing permissions and
 *  limitations under the License.
 *
 *=========================================================================*/


#include <hip/hip_runtime.h>
#include <stdio.h>

/** This is for parallel reduction and is modified from NVIDIA's example code */

// Utility class used to avoid linker errors with extern
// unsized shared memory arrays with templated type
template<class T>
struct SharedMemory
{
    __device__ inline operator T *()
    {
        extern __shared__ int __smem[];
        return (T *)__smem;
    }

    __device__ inline operator const T *() const
    {
        extern __shared__ int __smem[];
        return (T *)__smem;
    }
};

// specialize for double to avoid unaligned memory
// access compile errors
template<>
struct SharedMemory<double>
{
    __device__ inline operator double *()
    {
        extern __shared__ double __smem_d[];
        return (double *)__smem_d;
    }

    __device__ inline operator const double *() const
    {
        extern __shared__ double __smem_d[];
        return (double *)__smem_d;
    }
};

// The following defines are set during runtime compilation, see reduction.cpp
// #define T float
// #define blockSize 128
// #define nIsPow2 1
/*
    This version is completely unrolled.  It uses a template parameter to achieve
    optimal code for any (power of 2) number of threads.  This requires a switch
    statement in the host code to handle all the different thread block sizes at
    compile time.

    Note, this kernel needs a minimum of 64*sizeof(T) bytes of shared memory.
    In other words if blockSize <= 32, allocate 64*sizeof(T) bytes.
    If blockSize > 32, allocate blockSize*sizeof(T) bytes.
*/
template <class T>
__device__ void reduce5(T *g_idata, T *g_odata, unsigned int n, unsigned int blockSize)
{
    T *sdata = SharedMemory<T>();

    // perform first level of reduction,
    // reading from global memory, writing to shared memory
    unsigned int tid = threadIdx.x;
    unsigned int i = blockIdx.x * (blockSize * 2) + threadIdx.x;

    T mySum = (i < n) ? g_idata[i] : 0;

    if (i + blockSize < n)
        mySum += g_idata[i+blockSize];

    sdata[tid] = mySum;
    __syncthreads();

    // do reduction in shared mem
    if (blockSize >= 512) { if (tid < 256) { sdata[tid] = mySum = mySum + sdata[tid + 256]; } __syncthreads(); }
    if (blockSize >= 256) { if (tid < 128) { sdata[tid] = mySum = mySum + sdata[tid + 128]; } __syncthreads(); }
    if (blockSize >= 128) { if (tid <  64) { sdata[tid] = mySum = mySum + sdata[tid +  64]; } __syncthreads(); }

    if (tid < 32)
    {
        // now that we are using warp-synchronous programming (below)
        // we need to declare our shared memory volatile so that the compiler
        // doesn't reorder stores to it and induce incorrect behavior.
        volatile T *smem = sdata;
        if (blockSize >=  64) { smem[tid] = mySum = mySum + smem[tid + 32]; }
        if (blockSize >=  32) { smem[tid] = mySum = mySum + smem[tid + 16]; }
        if (blockSize >=  16) { smem[tid] = mySum = mySum + smem[tid +  8]; }
        if (blockSize >=   8) { smem[tid] = mySum = mySum + smem[tid +  4]; }
        if (blockSize >=   4) { smem[tid] = mySum = mySum + smem[tid +  2]; }
        if (blockSize >=   2) { smem[tid] = mySum = mySum + smem[tid +  1]; }
    }

    // write result for this block to global mem
    if (tid == 0) g_odata[blockIdx.x] = sdata[0];
}

/*
    This version adds multiple elements per thread sequentially.  This reduces the overall
    cost of the algorithm while keeping the work complexity O(n) and the step complexity O(log n).
    (Brent's Theorem optimization)

    Note, this kernel needs a minimum of 64*sizeof(T) bytes of shared memory.
    In other words if blockSize <= 32, allocate 64*sizeof(T) bytes.
    If blockSize > 32, allocate blockSize*sizeof(T) bytes.
*/
template <class T>
__device__ void reduce6(T *g_idata, T *g_odata, unsigned int n, unsigned int blockSize, bool nIsPow2)
{
    T *sdata = SharedMemory<T>();

    // perform first level of reduction,
    // reading from global memory, writing to shared memory
    unsigned int tid = threadIdx.x;
    unsigned int i = blockIdx.x * (blockSize * 2) + threadIdx.x;
    unsigned int gridSize = blockSize*2*gridDim.x;

    T mySum = 0;

    // we reduce multiple elements per thread.  The number is determined by the
    // number of active thread blocks (via gridDim).  More blocks will result
    // in a larger gridSize and therefore fewer elements per thread
    while (i < n)
    {
        mySum += g_idata[i];

        // ensure we don't read out of bounds -- this is optimized away for powerOf2 sized arrays
        if (nIsPow2 || i + blockSize < n)
            mySum += g_idata[i+blockSize];

        i += gridSize;
    }

    // each thread puts its local sum into shared memory
    sdata[tid] = mySum;
    __syncthreads();

    // do reduction in shared mem
    if (blockSize >= 512) { if (tid < 256) { sdata[tid] = mySum = mySum + sdata[tid + 256]; } __syncthreads(); }
    if (blockSize >= 256) { if (tid < 128) { sdata[tid] = mySum = mySum + sdata[tid + 128]; } __syncthreads(); }
    if (blockSize >= 128) { if (tid <  64) { sdata[tid] = mySum = mySum + sdata[tid +  64]; } __syncthreads(); }

    if (tid < 32)
    {
        // now that we are using warp-synchronous programming (below)
        // we need to declare our shared memory volatile so that the compiler
        // doesn't reorder stores to it and induce incorrect behavior.
        volatile T *smem = sdata;

        if (blockSize >=  64) { smem[tid] = mySum = mySum + smem[tid + 32]; }
        if (blockSize >=  32) { smem[tid] = mySum = mySum + smem[tid + 16]; }
        if (blockSize >=  16) { smem[tid] = mySum = mySum + smem[tid +  8]; }
        if (blockSize >=   8) { smem[tid] = mySum = mySum + smem[tid +  4]; }
        if (blockSize >=   4) { smem[tid] = mySum = mySum + smem[tid +  2]; }
        if (blockSize >=   2) { smem[tid] = mySum = mySum + smem[tid +  1]; }
    }
    
    // write result for this block to global mem
    if (tid == 0)        
        g_odata[blockIdx.x] = sdata[0];
}

#define MAKE_reduce5_TEMPLATE_IMPL(_exportname_, T) \
  extern "C" __global__ void _exportname_(T* g_idata, T *g_odata, unsigned int n, unsigned int blockSize)\
  {\
    reduce5<T>(g_idata, g_odata, n, blockSize);\
  }

#define MAKE_reduce6_TEMPLATE_IMPL(_exportname_, T) \
  extern "C" __global__ void _exportname_(T* g_idata, T *g_odata, unsigned int n, unsigned int blockSize, bool nIsPow2)\
  {\
    reduce6<T>(g_idata, g_odata, n, blockSize, nIsPow2);\
  }

#define MAKE_IMPL(_func_)\
  MAKE_##_func_##_TEMPLATE_IMPL(_func_##_c, char);\
  MAKE_##_func_##_TEMPLATE_IMPL(_func_##_uc, unsigned char);\
  MAKE_##_func_##_TEMPLATE_IMPL(_func_##_s, short);\
  MAKE_##_func_##_TEMPLATE_IMPL(_func_##_i, int);\
  MAKE_##_func_##_TEMPLATE_IMPL(_func_##_ui, unsigned int);\
  MAKE_##_func_##_TEMPLATE_IMPL(_func_##_f, float);\
  MAKE_##_func_##_TEMPLATE_IMPL(_func_##_d, double);

// Export the template implementations
MAKE_IMPL(reduce5);
MAKE_IMPL(reduce6);
